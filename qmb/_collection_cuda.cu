#include "hip/hip_runtime.h"
#include <c10/cuda/CUDAStream.h>
#include <hip/hip_runtime.h>
#include <thrust/binary_search.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <torch/extension.h>

namespace qmb_collection_cuda {

constexpr torch::DeviceType device = torch::kCUDA;

template<int value>
struct ConstInt {
    static constexpr int get_value() {
        return value;
    }
};

template<typename R>
R to_const_int_helper(int value) {
    TORCH_CHECK(false, "dimension not allowed");
}

template<typename R, int Head, int... Tail>
R to_const_int_helper(int value) {
    if (value == Head) {
        return ConstInt<Head>();
    } else {
        return to_const_int_helper<R, Tail...>(value);
    }
}

template<int... Values>
auto to_const_int(int value, std::integer_sequence<int, Values...>) {
    using R = std::variant<ConstInt<Values>...>;
    return to_const_int_helper<R, Values...>(value);
}

template<typename T, int size>
struct array_less {
    __host__ __device__ bool operator()(const std::array<T, size>& lhs, const std::array<T, size>& rhs) const {
        for (auto i = 0; i < size; ++i) {
            if (lhs[i] < rhs[i]) {
                return true;
            }
            if (lhs[i] > rhs[i]) {
                return false;
            }
        }
        return false;
    }
};

template<typename T, int size>
struct array_equal {
    __host__ __device__ bool operator()(const std::array<T, size>& lhs, const std::array<T, size>& rhs) const {
        for (auto i = 0; i < size; ++i) {
            if (lhs[i] != rhs[i]) {
                return false;
            }
        }
        return true;
    }
};

template<typename T, int size>
struct array_reduce {
    __host__ __device__ std::array<T, size> operator()(const std::array<T, size>& lhs, const std::array<T, size>& rhs) const {
        std::array<T, size> result;
        for (auto i = 0; i < size; ++i) {
            result[i] = lhs[i] + rhs[i];
        }
        return result;
    }
};

template<typename T, int size>
struct array_square_greater {
    __host__ __device__ T square(const std::array<T, size>& value) const {
        T result = 0;
        for (auto i = 0; i < size; ++i) {
            result += value[i] * value[i];
        }
        return result;
    }
    __host__ __device__ bool operator()(const std::array<T, size>& lhs, const std::array<T, size>& rhs) const {
        return square(lhs) > square(rhs);
    }
};

template<int n_qubits, int n_values>
void sort_impl(torch::Tensor& key, torch::Tensor& value) {
    std::int64_t length = key.size(0);
    thrust::sort_by_key(
        thrust::device.on(at::cuda::getCurrentCUDAStream(key.device().index())),
        reinterpret_cast<std::array<std::uint8_t, n_qubits>*>(key.data_ptr()),
        reinterpret_cast<std::array<std::uint8_t, n_qubits>*>(key.data_ptr()) + length,
        reinterpret_cast<std::array<double, n_values>*>(value.data_ptr()),
        array_less<std::uint8_t, n_qubits>()
    );
}

template<typename NQubits, typename NValues>
void sort(int n_qubits, int n_values, torch::Tensor& key, torch::Tensor& value) {
    std::visit(
        [&](auto n_qubits_handle, auto n_values_handle) {
            constexpr int n_qubits = n_qubits_handle.get_value();
            constexpr int n_values = n_values_handle.get_value();
            sort_impl<n_qubits, n_values>(key, value);
        },
        to_const_int(n_qubits, NQubits()),
        to_const_int(n_values, NValues())
    );
}

template<int n_qubits, int n_values>
void merge_impl(
    const torch::Tensor& key_1,
    const torch::Tensor& value_1,
    const torch::Tensor& key_2,
    const torch::Tensor& value_2,
    torch::Tensor& key_result,
    torch::Tensor& value_result
) {
    std::int64_t length_1 = key_1.size(0);
    std::int64_t length_2 = key_2.size(0);
    thrust::merge_by_key(
        thrust::device.on(at::cuda::getCurrentCUDAStream(key_1.device().index())),
        reinterpret_cast<std::array<std::uint8_t, n_qubits>*>(key_1.data_ptr()),
        reinterpret_cast<std::array<std::uint8_t, n_qubits>*>(key_1.data_ptr()) + length_1,
        reinterpret_cast<std::array<std::uint8_t, n_qubits>*>(key_2.data_ptr()),
        reinterpret_cast<std::array<std::uint8_t, n_qubits>*>(key_2.data_ptr()) + length_2,
        reinterpret_cast<std::array<double, n_values>*>(value_1.data_ptr()),
        reinterpret_cast<std::array<double, n_values>*>(value_2.data_ptr()),
        reinterpret_cast<std::array<std::uint8_t, n_qubits>*>(key_result.data_ptr()),
        reinterpret_cast<std::array<double, n_values>*>(value_result.data_ptr()),
        array_less<std::uint8_t, n_qubits>()
    );
}

template<typename NQubits, typename NValues>
void merge(
    int n_qubits,
    int n_values,
    const torch::Tensor& key_1,
    const torch::Tensor& value_1,
    const torch::Tensor& key_2,
    const torch::Tensor& value_2,
    torch::Tensor& key_result,
    torch::Tensor& value_result
) {
    std::visit(
        [&](auto n_qubits_handle, auto n_values_handle) {
            constexpr int n_qubits = n_qubits_handle.get_value();
            constexpr int n_values = n_values_handle.get_value();
            merge_impl<n_qubits, n_values>(key_1, value_1, key_2, value_2, key_result, value_result);
        },
        to_const_int(n_qubits, NQubits()),
        to_const_int(n_values, NValues())
    );
}

template<int n_qubits, int n_values>
std::int64_t reduce_impl(const torch::Tensor& key, const torch::Tensor& value, torch::Tensor& key_result, torch::Tensor& value_result) {
    std::int64_t length = key.size(0);
    auto [key_end, value_end] = thrust::reduce_by_key(
        thrust::device.on(at::cuda::getCurrentCUDAStream(key.device().index())),
        reinterpret_cast<std::array<std::uint8_t, n_qubits>*>(key.data_ptr()),
        reinterpret_cast<std::array<std::uint8_t, n_qubits>*>(key.data_ptr()) + length,
        reinterpret_cast<std::array<double, n_values>*>(value.data_ptr()),
        reinterpret_cast<std::array<std::uint8_t, n_qubits>*>(key_result.data_ptr()),
        reinterpret_cast<std::array<double, n_values>*>(value_result.data_ptr()),
        array_equal<std::uint8_t, n_qubits>(),
        array_reduce<double, n_values>()
    );
    return key_end - reinterpret_cast<std::array<std::uint8_t, n_qubits>*>(key_result.data_ptr());
}

template<typename NQubits, typename NValues>
std::int64_t
reduce(int n_qubits, int n_values, const torch::Tensor& key, const torch::Tensor& value, torch::Tensor& key_result, torch::Tensor& value_result) {
    return std::visit(
        [&](auto n_qubits_handle, auto n_values_handle) {
            constexpr int n_qubits = n_qubits_handle.get_value();
            constexpr int n_values = n_values_handle.get_value();
            return reduce_impl<n_qubits, n_values>(key, value, key_result, value_result);
        },
        to_const_int(n_qubits, NQubits()),
        to_const_int(n_values, NValues())
    );
}

template<int n_qubits, int n_values>
__global__ void
ensure_kernel(std::int64_t length, std::int64_t length_config, const std::array<std::uint8_t, n_qubits>* key, std::array<double, n_values>* value) {
    std::int64_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= length_config) {
        return;
    }
    std::int64_t low = 0;
    std::int64_t high = length - 1;
    std::int64_t mid = 0;
    auto compare = array_less<std::uint8_t, n_qubits>();
    while (low <= high) {
        mid = (low + high) / 2;
        if (compare(key[length_config + mid], key[i])) {
            low = mid + 1;
        } else if (compare(key[i], key[length_config + mid])) {
            high = mid - 1;
        } else {
            for (auto j = 0; j < n_values; ++j) {
                value[i][j] = value[length_config + mid][j];
                value[length_config + mid][j] = 0;
            }
            return;
        }
    }
}

template<int n_qubits, int n_values>
std::int64_t ensure_impl(torch::Tensor& key, torch::Tensor& value, std::int64_t length_config) {
    std::int64_t length = key.size(0) - length_config;

    std::int64_t device_id = key.device().index();
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device_id);
    std::int64_t threads_per_block = prop.maxThreadsPerBlock;
    std::int64_t num_blocks = (length_config + threads_per_block - 1) / threads_per_block;

    ensure_kernel<n_qubits, n_values><<<num_blocks, threads_per_block, 0, at::cuda::getCurrentCUDAStream(device_id)>>>(
        length,
        length_config,
        reinterpret_cast<std::array<std::uint8_t, n_qubits>*>(key.data_ptr()),
        reinterpret_cast<std::array<double, n_values>*>(value.data_ptr())
    );

    thrust::sort_by_key(
        thrust::device.on(at::cuda::getCurrentCUDAStream(device_id)),
        reinterpret_cast<std::array<double, n_values>*>(value.data_ptr()) + length_config,
        reinterpret_cast<std::array<double, n_values>*>(value.data_ptr()) + length_config + length,
        reinterpret_cast<std::array<std::uint8_t, n_qubits>*>(key.data_ptr()) + length_config,
        array_square_greater<double, n_values>()
    );

    std::array<double, n_values> zero_array;
    auto end = thrust::lower_bound(
        thrust::device.on(at::cuda::getCurrentCUDAStream(device_id)),
        reinterpret_cast<std::array<double, n_values>*>(value.data_ptr()) + length_config,
        reinterpret_cast<std::array<double, n_values>*>(value.data_ptr()) + length_config + length,
        zero_array,
        array_square_greater<double, n_values>()
    );
    return end - reinterpret_cast<std::array<double, n_values>*>(value.data_ptr());
}

template<typename NQubits, typename NValues>
std::int64_t ensure(int n_qubits, int n_values, torch::Tensor& key, torch::Tensor& value, std::int64_t length_config) {
    return std::visit(
        [&](auto n_qubits_handle, auto n_values_handle) {
            constexpr int n_qubits = n_qubits_handle.get_value();
            constexpr int n_values = n_values_handle.get_value();
            return ensure_impl<n_qubits, n_values>(key, value, length_config);
        },
        to_const_int(n_qubits, NQubits()),
        to_const_int(n_values, NValues())
    );
}

// key: A uint8 tensor of shape [length_x, n_qubits]
// value: A float64 tensor of shape [length_x, n_values] where n_values = 1 or 2.

template<typename NQubits, typename NValues>
auto sort_interface(torch::Tensor& key, torch::Tensor& value) -> std::tuple<torch::Tensor, torch::Tensor> {
    std::int64_t length = key.size(0);
    std::int64_t n_qubits = key.size(1);
    std::int64_t n_values = value.size(1);

    TORCH_CHECK(key.is_contiguous(), "key must be contiguous");
    TORCH_CHECK(value.is_contiguous(), "value must be contiguous");
    TORCH_CHECK(key.ndimension() == 2, "key must be a 2D tensor");
    TORCH_CHECK(value.ndimension() == 2, "value must be a 2D tensor");
    TORCH_CHECK(key.size(0) == value.size(0), "key and value must have the same length");

    sort<NQubits, NValues>(n_qubits, n_values, key, value);

    return std::make_tuple(key, value);
}

template<typename NQubits, typename NValues>
auto merge_interface(const torch::Tensor& key_1, const torch::Tensor& value_1, const torch::Tensor& key_2, const torch::Tensor& value_2)
    -> std::tuple<torch::Tensor, torch::Tensor> {
    std::int64_t length_1 = key_1.size(0);
    std::int64_t length_2 = key_2.size(0);
    std::int64_t n_qubits = key_1.size(1);
    std::int64_t n_values = value_1.size(1);

    TORCH_CHECK(key_1.is_contiguous(), "key_1 must be contiguous");
    TORCH_CHECK(key_2.is_contiguous(), "key_2 must be contiguous");
    TORCH_CHECK(value_1.is_contiguous(), "value_1 must be contiguous");
    TORCH_CHECK(value_2.is_contiguous(), "value_2 must be contiguous");
    TORCH_CHECK(key_1.ndimension() == 2, "key_1 must be a 2D tensor");
    TORCH_CHECK(key_1.size(0) == length_1, "key_1 must have the correct length");
    TORCH_CHECK(key_1.size(1) == n_qubits, "key_1 must have the correct length");
    TORCH_CHECK(value_1.ndimension() == 2, "value_1 must be a 2D tensor");
    TORCH_CHECK(value_1.size(0) == length_1, "value_1 must have the correct length");
    TORCH_CHECK(value_1.size(1) == n_values, "value_1 must have the correct length");
    TORCH_CHECK(key_2.ndimension() == 2, "key_2 must be a 2D tensor");
    TORCH_CHECK(key_2.size(0) == length_2, "key_2 must have the correct length");
    TORCH_CHECK(key_2.size(1) == n_qubits, "key_2 must have the correct length");
    TORCH_CHECK(value_2.ndimension() == 2, "value_2 must be a 2D tensor");
    TORCH_CHECK(value_2.size(0) == length_2, "value_2 must have the correct length");
    TORCH_CHECK(value_2.size(1) == n_values, "value_2 must have the correct length");

    std::int64_t device_id = key_1.device().index();
    auto key_result = torch::empty({length_1 + length_2, n_qubits}, torch::TensorOptions().dtype(torch::kUInt8).device(device, device_id));
    auto value_result = torch::empty({length_1 + length_2, n_values}, torch::TensorOptions().dtype(torch::kFloat64).device(device, device_id));

    merge<NQubits, NValues>(n_qubits, n_values, key_1, value_1, key_2, value_2, key_result, value_result);

    return std::make_tuple(key_result, value_result);
}

template<typename NQubits, typename NValues>
auto reduce_interface(const torch::Tensor& key, const torch::Tensor& value) -> std::tuple<torch::Tensor, torch::Tensor> {
    std::int64_t length = key.size(0);
    std::int64_t n_qubits = key.size(1);
    std::int64_t n_values = value.size(1);

    TORCH_CHECK(key.is_contiguous(), "key must be contiguous");
    TORCH_CHECK(value.is_contiguous(), "value must be contiguous");
    TORCH_CHECK(key.ndimension() == 2, "key must be a 2D tensor");
    TORCH_CHECK(value.ndimension() == 2, "value must be a 2D tensor");
    TORCH_CHECK(key.size(0) == value.size(0), "key and value must have the same length");

    std::int64_t device_id = key.device().index();
    auto key_result = torch::empty({length, n_qubits}, torch::TensorOptions().dtype(torch::kUInt8).device(device, device_id));
    auto value_result = torch::empty({length, n_values}, torch::TensorOptions().dtype(torch::kFloat64).device(device, device_id));

    std::int64_t size = reduce<NQubits, NValues>(n_qubits, n_values, key, value, key_result, value_result);
    auto slice = torch::indexing::Slice(torch::indexing::None, size);

    return std::make_tuple(key_result.index({slice}), value_result.index({slice}));
}

template<typename NQubits, typename NValues>
auto ensure_interface(torch::Tensor& key, torch::Tensor& value, std::int64_t length_config) -> std::tuple<torch::Tensor, torch::Tensor> {
    std::int64_t length = key.size(0);
    std::int64_t n_qubits = key.size(1);
    std::int64_t n_values = value.size(1);

    TORCH_CHECK(key.is_contiguous(), "key must be contiguous");
    TORCH_CHECK(value.is_contiguous(), "value must be contiguous");
    TORCH_CHECK(key.ndimension() == 2, "key must be a 2D tensor");
    TORCH_CHECK(value.ndimension() == 2, "value must be a 2D tensor");
    TORCH_CHECK(key.size(0) == value.size(0), "key and value must have the same length");

    std::int64_t size = ensure<NQubits, NValues>(n_qubits, n_values, key, value, length_config);
    auto slice = torch::indexing::Slice(torch::indexing::None, size);

    return std::make_tuple(key.index({slice}), value.index({slice}));
}

#ifndef NQUBYTES
#define NQUBYTES 0
#endif

#ifndef QMB_LIBRARY_HELPER
#define QMB_LIBRARY_HELPER(x) qmb_collection_##x
#endif
#ifndef QMB_LIBRARY
#define QMB_LIBRARY(x) QMB_LIBRARY_HELPER(x)
#endif

#if NQUBYTES != 0
TORCH_LIBRARY_IMPL(QMB_LIBRARY(NQUBYTES), CUDA, m) {
    m.impl("sort_", sort_interface<std::integer_sequence<int, NQUBYTES>, std::integer_sequence<int, 1, 2>>);
    m.impl("merge", merge_interface<std::integer_sequence<int, NQUBYTES>, std::integer_sequence<int, 1, 2>>);
    m.impl("reduce", reduce_interface<std::integer_sequence<int, NQUBYTES>, std::integer_sequence<int, 1, 2>>);
    m.impl("ensure_", ensure_interface<std::integer_sequence<int, NQUBYTES>, std::integer_sequence<int, 1, 2>>);
}
#endif

} // namespace qmb_collection_cuda
