#include "hip/hip_runtime.h"
// This file implements a PyTorch operator designed to efficiently iterate over Hamiltonian terms in quantum many-body systems on CUDA devices.
// It is tailored to support both fermion and boson systems, with specific optimizations for different particle cutoffs.
// It utilizes several template arguments to tailor the computation:
//   - max_op_number: Specifies the maximum number of operations for all terms in the Hamiltonian, typically set to 4.
//   - particle_cut: Determines the system type; particle_cut >= 2 indicates a boson system with a specific number cut,
//                   while particle_cut = 1 signifies a fermion system.
// This file encompasses multiple functions designed to achieve the following objectives:
// 1. `search_kernel`: A device function responsible for processing a single term and a single configuration within the Hamiltonian.
// 2. `search_kernel_interface`: A global function that orchestrates the invocation of `search_kernel`. It determines which term and configuration
//    each thread should process based on the thread and grid indices.
// 3. `launch_search_kernel`: A host function dedicated to launching the `search_kernel_interface`. It strategically allocates grid and thread
//    dimensions to ensure all terms and configurations are processed efficiently.
// 4. `python_interface`: The PyTorch operator interface, which integrates the CUDA kernels into the PyTorch framework.

#include <c10/cuda/CUDAStream.h>
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <torch/extension.h>

namespace qmb_hamiltonian_cuda {

constexpr torch::DeviceType device = torch::kCUDA;

template<typename T, int size>
struct array_less {
    __host__ __device__ bool operator()(const std::array<T, size>& lhs, const std::array<T, size>& rhs) const {
        for (auto i = 0; i < size; ++i) {
            if (lhs[i] < rhs[i]) {
                return true;
            }
            if (lhs[i] > rhs[i]) {
                return false;
            }
        }
        return false;
    }
};

template<typename T, int size>
struct array_square_greater {
    __host__ __device__ T square(const std::array<T, size>& value) const {
        T result = 0;
        for (auto i = 0; i < size; ++i) {
            result += value[i] * value[i];
        }
        return result;
    }
    __host__ __device__ bool operator()(const std::array<T, size>& lhs, const std::array<T, size>& rhs) const {
        return square(lhs) > square(rhs);
    }
};

__device__ bool get_bit(std::uint8_t* data, int index) {
    return ((*data) & (1 << index)) >> index;
}

__device__ bool set_bit(std::uint8_t* data, int index, bool value) {
    if (value) {
        *data |= (1 << index);
    } else {
        *data &= ~(1 << index);
    }
}

template<std::int64_t max_op_number, std::int64_t n_qubytes, std::int64_t particle_cut>
__device__ std::pair<bool, bool> hamiltonian_apply_kernel(
    std::array<std::uint8_t, n_qubytes>& current_configs,
    std::int64_t term_index,
    std::int64_t batch_index,
    const std::array<std::int16_t, max_op_number>* site, // term_number
    const std::array<std::uint8_t, max_op_number>* kind // term_number
) {
    static_assert(particle_cut == 1 || particle_cut == 2, "particle_cut != 1 or 2 not implemented");
    bool success = true;
    bool parity = false;
    for (auto op_index = max_op_number; op_index-- > 0;) {
        auto site_single = site[term_index][op_index];
        auto kind_single = kind[term_index][op_index];
        if (kind_single == 2) {
            continue;
        }
        auto to_what = kind_single;
        if (get_bit(&current_configs[site_single / 8], site_single % 8) == to_what) {
            success = false;
            break;
        }
        set_bit(&current_configs[site_single / 8], site_single % 8, to_what);
        if constexpr (particle_cut == 1) {
            for (auto s = 0; s < site_single; ++s) {
                parity ^= get_bit(&current_configs[s / 8], s % 8);
            }
        }
    }
    return std::make_pair(success, parity);
}

template<std::int64_t max_op_number, std::int64_t n_qubytes, std::int64_t particle_cut>
__device__ void apply_within_kernel(
    std::int64_t term_index,
    std::int64_t batch_index,
    std::int64_t term_number,
    std::int64_t batch_size,
    std::int64_t result_batch_size,
    const std::array<std::int16_t, max_op_number>* site, // term_number
    const std::array<std::uint8_t, max_op_number>* kind, // term_number
    const std::array<double, 2>* coef, // term_number
    const std::array<std::uint8_t, n_qubytes>* configs, // batch_size
    const std::array<double, 2>* psi, // batch_size
    const std::array<std::uint8_t, n_qubytes>* result_configs, // result_batch_size
    std::array<double, 2>* result_psi
) {
    std::array<std::uint8_t, n_qubytes> current_configs = configs[batch_index];
    auto [success, parity] = hamiltonian_apply_kernel<max_op_number, n_qubytes, particle_cut>(current_configs, term_index, batch_index, site, kind);

    if (!success) {
        return;
    }
    success = false;
    std::int64_t low = 0;
    std::int64_t high = result_batch_size - 1;
    std::int64_t mid = 0;
    auto compare = array_less<std::uint8_t, n_qubytes>();
    while (low <= high) {
        mid = (low + high) / 2;
        if (compare(current_configs, result_configs[mid])) {
            high = mid - 1;
        } else if (compare(result_configs[mid], current_configs)) {
            low = mid + 1;
        } else {
            success = true;
            break;
        }
    }
    if (!success) {
        return;
    }
    std::int8_t sign = parity ? -1 : +1;
    atomicAdd(&result_psi[mid][0], sign * (coef[term_index][0] * psi[batch_index][0] - coef[term_index][1] * psi[batch_index][1]));
    atomicAdd(&result_psi[mid][1], sign * (coef[term_index][0] * psi[batch_index][1] + coef[term_index][1] * psi[batch_index][0]));
}

template<std::int64_t max_op_number, std::int64_t n_qubytes, std::int64_t particle_cut>
__global__ void apply_within_kernel_interface(
    std::int64_t term_number,
    std::int64_t batch_size,
    std::int64_t result_batch_size,
    const std::array<std::int16_t, max_op_number>* site, // term_number
    const std::array<std::uint8_t, max_op_number>* kind, // term_number
    const std::array<double, 2>* coef, // term_number
    const std::array<std::uint8_t, n_qubytes>* configs, // batch_size
    const std::array<double, 2>* psi, // batch_size
    const std::array<std::uint8_t, n_qubytes>* result_configs, // result_batch_size
    std::array<double, 2>* result_psi
) {
    int term_index = blockIdx.x * blockDim.x + threadIdx.x;
    int batch_index = blockIdx.y * blockDim.y + threadIdx.y;

    if (term_index < term_number && batch_index < batch_size) {
        apply_within_kernel<max_op_number, n_qubytes, particle_cut>(
            term_index,
            batch_index,
            term_number,
            batch_size,
            result_batch_size,
            site,
            kind,
            coef,
            configs,
            psi,
            result_configs,
            result_psi
        );
    }
}

template<std::int64_t max_op_number, std::int64_t n_qubytes, std::int64_t particle_cut>
auto apply_within_interface(
    const torch::Tensor& configs,
    const torch::Tensor& psi,
    const torch::Tensor& result_configs,
    const torch::Tensor& site,
    const torch::Tensor& kind,
    const torch::Tensor& coef
) -> torch::Tensor {
    std::int64_t device_id = configs.device().index();
    std::int64_t batch_size = configs.size(0);
    std::int64_t result_batch_size = result_configs.size(0);
    std::int64_t term_number = site.size(0);

    auto stream = at::cuda::getCurrentCUDAStream(device_id);
    auto policy = thrust::device.on(stream);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device_id);
    std::int64_t max_threads_per_block = prop.maxThreadsPerBlock;

    auto sorted_result_configs = result_configs.clone(torch::MemoryFormat::Contiguous);
    auto result_sort_index = torch::arange(result_batch_size, torch::TensorOptions().dtype(torch::kInt64).device(device, device_id));
    auto sorted_result_psi = torch::zeros({result_batch_size, 2}, torch::TensorOptions().dtype(torch::kDouble).device(device, device_id));

    thrust::sort_by_key(
        policy,
        reinterpret_cast<std::array<std::uint8_t, n_qubytes>*>(sorted_result_configs.data_ptr()),
        reinterpret_cast<std::array<std::uint8_t, n_qubytes>*>(sorted_result_configs.data_ptr()) + result_batch_size,
        reinterpret_cast<std::int64_t*>(result_sort_index.data_ptr()),
        array_less<std::uint8_t, n_qubytes>()
    );

    auto threads_per_block = dim3{1, max_threads_per_block >> 1}; // I don't know why, but need to divide by 2 to avoid errors
    auto num_blocks = dim3{
        (std::int32_t(term_number) + threads_per_block.x - 1) / threads_per_block.x,
        (std::int32_t(batch_size) + threads_per_block.y - 1) / threads_per_block.y
    };

    apply_within_kernel_interface<max_op_number, n_qubytes, particle_cut><<<num_blocks, threads_per_block, 0, stream>>>(
        term_number,
        batch_size,
        result_batch_size,
        reinterpret_cast<const std::array<std::int16_t, max_op_number>*>(site.data_ptr()),
        reinterpret_cast<const std::array<std::uint8_t, max_op_number>*>(kind.data_ptr()),
        reinterpret_cast<const std::array<double, 2>*>(coef.data_ptr()),
        reinterpret_cast<const std::array<std::uint8_t, n_qubytes>*>(configs.data_ptr()),
        reinterpret_cast<const std::array<double, 2>*>(psi.data_ptr()),
        reinterpret_cast<const std::array<std::uint8_t, n_qubytes>*>(sorted_result_configs.data_ptr()),
        reinterpret_cast<std::array<double, 2>*>(sorted_result_psi.data_ptr())
    );
    hipStreamSynchronize(stream);

    auto result_psi = torch::zeros_like(sorted_result_psi);
    result_psi.index_put_({result_sort_index}, sorted_result_psi);
    return result_psi;
}

constexpr std::uint64_t max_uint8_t = 256;
using largest_atomic_int = unsigned int;
using smallest_atomic_int = unsigned short int;

template<std::int64_t n_qubytes>
struct dictionary_tree {
    using child_t = dictionary_tree<n_qubytes - 1>;
    child_t* children[max_uint8_t];
    smallest_atomic_int exist[max_uint8_t];
    largest_atomic_int nonzero_count;

    __device__ bool add(std::uint8_t* begin, double real, double imag) {
        std::uint8_t index = *begin;
        if (children[index] == nullptr) {
            if (atomicCAS(&exist[index], smallest_atomic_int(0), smallest_atomic_int(1))) {
                while (atomicCAS((largest_atomic_int*)&children[index], largest_atomic_int(0), largest_atomic_int(0)) == 0) {
                }
            } else {
                auto new_child = (child_t*)malloc(sizeof(child_t));
                memset(new_child, 0, sizeof(child_t));
                children[index] = new_child;
                __threadfence();
            }
        }
        if (children[index]->add(begin + 1, real, imag)) {
            atomicAdd(&nonzero_count, 1);
            return true;
        } else {
            return false;
        }
    }

    template<std::int64_t n_total_qubytes>
    __device__ bool collect(std::uint64_t index, std::array<std::uint8_t, n_total_qubytes>* configs, std::array<double, 2>* psi) {
        std::uint64_t size_counter = 0;
        for (auto i = 0; i < max_uint8_t; ++i) {
            if (exist[i]) {
                std::uint64_t new_size_counter = size_counter + children[i]->nonzero_count;
                if (new_size_counter > index) {
                    std::uint64_t new_index = index - size_counter;
                    configs[index][n_total_qubytes - n_qubytes] = i;
                    bool empty = children[i]->collect<n_total_qubytes>(new_index, &configs[size_counter], &psi[size_counter]);
                    if (empty) {
                        free(children[i]);
                        children[i] = nullptr;
                        if (!atomicSub(&nonzero_count, 1)) {
                            return true;
                        }
                    }
                    return false;
                }
                size_counter += children[i]->nonzero_count;
            }
        }
        return !atomicSub(&nonzero_count, 1);
    }
};

template<>
struct dictionary_tree<1> {
    double values[max_uint8_t][2];
    smallest_atomic_int exist[max_uint8_t];
    largest_atomic_int nonzero_count;

    __device__ bool add(std::uint8_t* begin, double real, double imag) {
        std::uint8_t index = *begin;
        atomicAdd(&values[index][0], real);
        atomicAdd(&values[index][1], imag);
        if (atomicCAS(&exist[index], smallest_atomic_int(0), smallest_atomic_int(1))) {
            return false;
        } else {
            atomicAdd(&nonzero_count, 1);
            return true;
        }
    }

    template<std::int64_t n_total_qubytes>
    __device__ bool collect(std::uint64_t index, std::array<std::uint8_t, n_total_qubytes>* configs, std::array<double, 2>* psi) {
        std::uint64_t size_counter = 0;
        for (auto i = 0; i < max_uint8_t; ++i) {
            if (exist[i]) {
                if (size_counter == index) {
                    configs[index][n_total_qubytes - 1] = i;
                    psi[index][0] = values[i][0];
                    psi[index][1] = values[i][1];
                    return !atomicSub(&nonzero_count, 1);
                }
                size_counter += 1;
            }
        }
    }
};

template<std::int64_t max_op_number, std::int64_t n_qubytes, std::int64_t particle_cut>
__device__ void find_relative_kernel(
    std::int64_t term_index,
    std::int64_t batch_index,
    std::int64_t term_number,
    std::int64_t batch_size,
    const std::array<std::int16_t, max_op_number>* site, // term_number
    const std::array<std::uint8_t, max_op_number>* kind, // term_number
    const std::array<double, 2>* coef, // term_number
    const std::array<std::uint8_t, n_qubytes>* configs, // batch_size
    const std::array<double, 2>* psi, // batch_size
    dictionary_tree<n_qubytes>* result_tree
) {
    std::array<std::uint8_t, n_qubytes> current_configs = configs[batch_index];
    auto [success, parity] = hamiltonian_apply_kernel<max_op_number, n_qubytes, particle_cut>(current_configs, term_index, batch_index, site, kind);

    if (!success) {
        return;
    }
    success = true;
    std::int64_t low = 0;
    std::int64_t high = batch_size - 1;
    std::int64_t mid = 0;
    auto compare = array_less<std::uint8_t, n_qubytes>();
    while (low <= high) {
        mid = (low + high) / 2;
        if (compare(current_configs, configs[mid])) {
            high = mid - 1;
        } else if (compare(configs[mid], current_configs)) {
            low = mid + 1;
        } else {
            success = false;
            break;
        }
    }
    if (!success) {
        return;
    }
    std::int8_t sign = parity ? -1 : +1;
    result_tree->add(
        current_configs.data(),
        sign * (coef[term_index][0] * psi[batch_index][0] - coef[term_index][1] * psi[batch_index][1]),
        sign * (coef[term_index][0] * psi[batch_index][1] + coef[term_index][1] * psi[batch_index][0])
    );
}

template<std::int64_t max_op_number, std::int64_t n_qubytes, std::int64_t particle_cut>
__global__ void find_relative_kernel_interface(
    std::int64_t term_number,
    std::int64_t batch_size,
    const std::array<std::int16_t, max_op_number>* site, // term_number
    const std::array<std::uint8_t, max_op_number>* kind, // term_number
    const std::array<double, 2>* coef, // term_number
    const std::array<std::uint8_t, n_qubytes>* configs, // batch_size
    const std::array<double, 2>* psi, // batch_size
    dictionary_tree<n_qubytes>* result_tree
) {
    int term_index = blockIdx.x * blockDim.x + threadIdx.x;
    int batch_index = blockIdx.y * blockDim.y + threadIdx.y;

    if (term_index < term_number && batch_index < batch_size) {
        find_relative_kernel<max_op_number, n_qubytes, particle_cut>(
            term_index,
            batch_index,
            term_number,
            batch_size,
            site,
            kind,
            coef,
            configs,
            psi,
            result_tree
        );
    }
}

template<std::int64_t max_op_number, std::int64_t n_qubytes, std::int64_t particle_cut>
__global__ void collect_kernel_interface(
    std::uint64_t result_size,
    dictionary_tree<n_qubytes>* result_tree,
    std::array<std::uint8_t, n_qubytes>* configs,
    std::array<double, 2>* psi
) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < result_size) {
        result_tree->collect<n_qubytes>(index, configs, psi);
    }
}

template<std::int64_t max_op_number, std::int64_t n_qubytes, std::int64_t particle_cut>
auto find_relative_interface(
    const torch::Tensor& configs,
    const torch::Tensor& psi,
    const std::int64_t count_selected,
    const torch::Tensor& site,
    const torch::Tensor& kind,
    const torch::Tensor& coef
) -> torch::Tensor {
    std::int64_t device_id = configs.device().index();
    std::int64_t batch_size = configs.size(0);
    std::int64_t term_number = site.size(0);

    auto stream = at::cuda::getCurrentCUDAStream(device_id);
    auto policy = thrust::device.on(stream);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device_id);
    std::int64_t max_threads_per_block = prop.maxThreadsPerBlock;

    auto sorted_configs = configs.clone(torch::MemoryFormat::Contiguous);
    auto sorted_psi = psi.clone(torch::MemoryFormat::Contiguous);

    thrust::sort_by_key(
        policy,
        reinterpret_cast<std::array<std::uint8_t, n_qubytes>*>(sorted_configs.data_ptr()),
        reinterpret_cast<std::array<std::uint8_t, n_qubytes>*>(sorted_configs.data_ptr()) + batch_size,
        reinterpret_cast<std::array<double, 2>*>(sorted_psi.data_ptr()),
        array_less<std::uint8_t, n_qubytes>()
    );

    dictionary_tree<n_qubytes>* result_tree;
    hipMalloc(&result_tree, sizeof(dictionary_tree<n_qubytes>));
    hipMemset(result_tree, 0, sizeof(dictionary_tree<n_qubytes>));

    auto threads_per_block = dim3{1, max_threads_per_block >> 1}; // I don't know why, but need to divide by 2 to avoid errors
    auto num_blocks = dim3{
        (std::int32_t(term_number) + threads_per_block.x - 1) / threads_per_block.x,
        (std::int32_t(batch_size) + threads_per_block.y - 1) / threads_per_block.y
    };
    find_relative_kernel_interface<max_op_number, n_qubytes, particle_cut><<<num_blocks, threads_per_block, 0, stream>>>(
        term_number,
        batch_size,
        reinterpret_cast<const std::array<std::int16_t, max_op_number>*>(site.data_ptr()),
        reinterpret_cast<const std::array<std::uint8_t, max_op_number>*>(kind.data_ptr()),
        reinterpret_cast<const std::array<double, 2>*>(coef.data_ptr()),
        reinterpret_cast<const std::array<std::uint8_t, n_qubytes>*>(sorted_configs.data_ptr()),
        reinterpret_cast<const std::array<double, 2>*>(sorted_psi.data_ptr()),
        result_tree
    );
    hipStreamSynchronize(stream);

    largest_atomic_int result_size;
    hipMemcpy(&result_size, &result_tree->nonzero_count, sizeof(largest_atomic_int), hipMemcpyDeviceToHost);

    auto result_configs = torch::zeros({result_size, n_qubytes}, torch::TensorOptions().dtype(torch::kUInt8).device(device, device_id));
    auto result_psi = torch::zeros({result_size, 2}, torch::TensorOptions().dtype(torch::kDouble).device(device, device_id));

    auto threads_per_block_collect = max_threads_per_block >> 1;
    auto num_blocks_collect = (std::int32_t(result_size) + threads_per_block_collect - 1) / threads_per_block_collect;
    collect_kernel_interface<max_op_number, n_qubytes, particle_cut><<<num_blocks_collect, threads_per_block_collect, 0, stream>>>(
        result_size,
        result_tree,
        reinterpret_cast<std::array<std::uint8_t, n_qubytes>*>(result_configs.data_ptr()),
        reinterpret_cast<std::array<double, 2>*>(result_psi.data_ptr())
    );
    hipStreamSynchronize(stream);

    hipFree(result_tree);

    thrust::sort_by_key(
        policy,
        reinterpret_cast<std::array<double, 2>*>(result_psi.data_ptr()),
        reinterpret_cast<std::array<double, 2>*>(result_psi.data_ptr()) + result_size,
        reinterpret_cast<std::array<std::uint8_t, n_qubytes>*>(result_configs.data_ptr()),
        array_square_greater<double, 2>()
    );

    return result_configs.index({torch::indexing::Slice(torch::indexing::None, count_selected)});
}

#ifndef N_QUBYTES
#define N_QUBYTES 0
#endif
#ifndef PARTICLE_CUT
#define PARTICLE_CUT 0
#endif

#if N_QUBYTES != 0
#define QMB_LIBRARY_HELPER(x, y) qmb_hamiltonian_##x##_##y
#define QMB_LIBRARY(x, y) QMB_LIBRARY_HELPER(x, y)
TORCH_LIBRARY_IMPL(QMB_LIBRARY(N_QUBYTES, PARTICLE_CUT), CUDA, m) {
    m.impl("apply_within", apply_within_interface</*max_op_number=*/4, /*n_qubytes=*/N_QUBYTES, /*particle_cut=*/PARTICLE_CUT>);
    m.impl("find_relative", find_relative_interface</*max_op_number=*/4, /*n_qubytes=*/N_QUBYTES, /*particle_cut=*/PARTICLE_CUT>);
}
#undef QMB_LIBRARY
#undef QMB_LIBRARY_HELPER
#endif

} // namespace qmb_hamiltonian_cuda
